#include "hip/hip_runtime.h"
#include <objects/point/points.cuh>
#include <objects/point/kernels.cu>
#include <core/ImguiConfig.hpp>

void Points::init()
{
    isPing = true;
    //Generating buffers
    glGenBuffers (BUFFER_COUNT, BUFFER);     

    std::cout<<glGetError()<<std::endl;
    //init buffers
    uint *tempUint = new uint[MAX_NUM_POINTS];
    for (uint i = BUFFER_INDICES; i<= BUFFER_CELLID_PONG; i++)
    {
        glBindBuffer (GL_ARRAY_BUFFER, BUFFER[i]);
        glBufferData (GL_ARRAY_BUFFER, MAX_NUM_POINTS * sizeof(uint), tempUint, GL_DYNAMIC_DRAW);
    }
    delete []tempUint;

    float *tempFloat = new float[MAX_NUM_POINTS*3];
    for (uint i=BUFFER_POSITIONS_PING; i<=BUFFER_NORMALS; i++)
    {
        glBindBuffer( GL_ARRAY_BUFFER, BUFFER[i]);
        glBufferData (GL_ARRAY_BUFFER, MAX_NUM_POINTS * sizeof(float) * 3, tempFloat, GL_DYNAMIC_DRAW);
    }
    delete []tempFloat;
    for (uint i=BUFFER_DENSITIES_PING; i<=BUFFER_FREEZEPOINT; i++)
    {
        glBindBuffer( GL_ARRAY_BUFFER, BUFFER[i]);
        glBufferData (GL_ARRAY_BUFFER, MAX_NUM_POINTS * sizeof(float) , 0, GL_DYNAMIC_DRAW);
    }
    std::cout<<glGetError()<<std::endl;

    //Setting Ping VAO
    glGenVertexArrays (1, &pingVAO);
    glBindVertexArray (pingVAO);

    glBindBuffer( GL_ARRAY_BUFFER, BUFFER[BUFFER_POSITIONS_PING]);
    glEnableVertexAttribArray (0);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, NULL);

    glBindBuffer( GL_ARRAY_BUFFER, BUFFER[BUFFER_TEMP_PING]);
    glEnableVertexAttribArray (1);
    glVertexAttribPointer(1, 1, GL_FLOAT, GL_FALSE, 0, NULL);

    glBindBuffer( GL_ARRAY_BUFFER, BUFFER[BUFFER_COLORGRAD_PING]);
    glEnableVertexAttribArray (2);
    glVertexAttribPointer(2, 3, GL_FLOAT, GL_FALSE, 0, NULL);

    glBindBuffer( GL_ARRAY_BUFFER, BUFFER[BUFFER_DENSITIES_PING]);
    glEnableVertexAttribArray (3);
    glVertexAttribPointer(3, 1, GL_FLOAT, GL_FALSE, 0, NULL);


    //Setting Pong VAO;
    glGenVertexArrays (1, &pongVAO);
    glBindVertexArray (pongVAO);

    glBindBuffer( GL_ARRAY_BUFFER, BUFFER[BUFFER_POSITIONS_PONG]);
    glEnableVertexAttribArray (0);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, NULL);

    glBindBuffer( GL_ARRAY_BUFFER, BUFFER[BUFFER_TEMP_PONG]);
    glEnableVertexAttribArray (1);
    glVertexAttribPointer(1, 1, GL_FLOAT, GL_FALSE, 0, NULL);

    glBindBuffer( GL_ARRAY_BUFFER, BUFFER[BUFFER_COLORGRAD_PONG]);
    glEnableVertexAttribArray (2);
    glVertexAttribPointer(2, 3, GL_FLOAT, GL_FALSE, 0, NULL);

    glBindBuffer( GL_ARRAY_BUFFER, BUFFER[BUFFER_DENSITIES_PONG]);
    glEnableVertexAttribArray (3);
    glVertexAttribPointer(3, 1, GL_FLOAT, GL_FALSE, 0, NULL);


    //load temperature texture
    loadTempText();

    //Register CUDA resources
    for (int i=0; i<BUFFER_COUNT; i++)
        gpuErrchk( hipGraphicsGLRegisterBuffer(&(resources[i]), BUFFER[i], hipGraphicsRegisterFlagsWriteDiscard));
    //allocate space for constants

    //Allocate and init grid
    gpuErrchk(hipMalloc((void**)&(grid.gridRes), sizeof(uint3)));
    gpuErrchk(hipMalloc((void**)&(grid.startPoint), sizeof(float3)));
    gpuErrchk(hipMalloc((void**)&(grid.cellSize), sizeof(float3)));
    
    grid.gridRes = make_uint3(16, 16, 16);
    grid.startPoint = make_float3(-15, -15, -15);
    grid.cellSize = make_float3( -grid.startPoint.x*2/grid.gridRes.x, -grid.startPoint.y*2/grid.gridRes.y, -grid.startPoint.z*2/grid.gridRes.z);


    gpuErrchk(hipMalloc((void**)&(grid.count), sizeof(uint)*grid.gridRes.x*grid.gridRes.y*grid.gridRes.z));
    gpuErrchk(hipMalloc((void**)&(grid.firstIdx), sizeof(uint)*grid.gridRes.x*grid.gridRes.y*grid.gridRes.z));
    
    gpuErrchk(hipMalloc((void**)&dSumArray, sizeof(uint)
                *(grid.gridRes.x * grid.gridRes.y * grid.gridRes.z -1)/2048+1));  //dead param

    gpuErrchk(hipMalloc((void**)&dDensSum, sizeof(float)));
    gpuErrchk(hipMalloc((void**)&dDensCount, sizeof(int)));

    gpuErrchk(hipMalloc((void**)&tempPos, sizeof(float)*3*MAX_NUM_POINTS));



}

void Points::setValues ( const GLfloat *val, const GLuint &num, const GLuint &size, const int &bufferIdx)
{
    numPoints = num;
    gpuErrchk(hipMemcpy(dNumPoints, &numPoints, sizeof(uint), hipMemcpyHostToDevice));
    if ( numPoints > MAX_NUM_POINTS)
        exit(-1);
    glBindBuffer(GL_ARRAY_BUFFER, BUFFER[bufferIdx]);
    glBufferSubData(GL_ARRAY_BUFFER, 0, num*size*sizeof(GLfloat), val);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
}

void Points::addPars( const GLfloat *pos, const GLfloat *temp, const GLfloat *velos, const GLuint &num, const GLuint *s)
{
    if ( numPoints+num >= MAX_NUM_POINTS)
    {
        std::cout<<"To many particles\n";
        exit(-1);
    }

    int BUFFER_POSITIONS = 0;
    int BUFFER_TEMP = 0;
    int BUFFER_STATES = 0;
    int BUFFER_VELOCITIES = 0;
    if (isPing)
    {
        BUFFER_POSITIONS = BUFFER_POSITIONS_PING;
        BUFFER_TEMP = BUFFER_TEMP_PING;
        BUFFER_STATES = BUFFER_STATES_PING;
        BUFFER_VELOCITIES = BUFFER_VELOCITIES_PING;
    }
    else
    {
        BUFFER_POSITIONS = BUFFER_POSITIONS_PONG;
        BUFFER_TEMP = BUFFER_TEMP_PONG;
        BUFFER_STATES = BUFFER_STATES_PONG;
        BUFFER_VELOCITIES = BUFFER_VELOCITIES_PONG;
    }

    glBindBuffer(GL_ARRAY_BUFFER, BUFFER[BUFFER_POSITIONS]);
    glBufferSubData(GL_ARRAY_BUFFER, 3*numPoints*sizeof(GLfloat), 3*num*sizeof(GLfloat), pos);

    glBindBuffer(GL_ARRAY_BUFFER, BUFFER[BUFFER_TEMP]);
    glBufferSubData(GL_ARRAY_BUFFER, numPoints*sizeof(GLfloat), num*sizeof(GLfloat), temp);

    glBindBuffer(GL_ARRAY_BUFFER, BUFFER[BUFFER_STATES]);
    glBufferSubData(GL_ARRAY_BUFFER, numPoints*sizeof(GLuint), num*sizeof(GLuint), s);

    //set velo to zero
    glBindBuffer(GL_ARRAY_BUFFER, BUFFER[BUFFER_VELOCITIES]);
    glBufferSubData(GL_ARRAY_BUFFER, 3*numPoints*sizeof(GLfloat), 3*num*sizeof(GLfloat), velos);

    glBindBuffer(GL_ARRAY_BUFFER, 0);
    numPoints += num;


}

void Points::update()
{
    runKernels();
    float GPUsum;
    int GPUcount;
    gpuErrchk(hipMemcpy(&GPUsum, dDensSum, sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(&GPUcount, dDensCount, sizeof(int), hipMemcpyDeviceToHost));
    surfaceDens = GPUsum / GPUcount;
}

void Points::draw()
{

    if (isPing)
        glBindVertexArray(pingVAO);
    else
        glBindVertexArray(pongVAO);
    glDrawArrays( GL_POINTS, 0, numPoints);
    glBindVertexArray(0);
    isPing = !isPing; //swap buffer
}

void Points::loadTempText()     //load texture with il
{

    //TODO: load image with other tools
    //
    ILuint imageID;				// Create an image ID as a ULuint
    ilInit();
    GLuint textureID;			// Create a texture ID as a GLuint
    ILboolean success;			// Create a flag to keep track of success/failure
    ILenum error;				// Create a flag to keep track of the IL error state
    ilGenImages(1, &imageID); 		// Generate the image ID
    ilBindImage(imageID); 			// Bind the image
    success = ilLoadImage("./assets/temperature.png"); 	// Load the image file
    // If we managed to load the image, then we can start to do things with it...
    if (success)
    {
        // If the image is flipped (i.e. upside-down and mirrored, flip it the right way up!)
        ILinfo ImageInfo;
        iluGetImageInfo(&ImageInfo);
        if (ImageInfo.Origin == IL_ORIGIN_UPPER_LEFT)
        {
            iluFlipImage();
        }

        // Convert the image into a suitable format to work with
        // NOTE: If your image contains alpha channel you can replace IL_RGB with IL_RGBA
        success = ilConvertImage(IL_RGB, IL_UNSIGNED_BYTE);

        // Quit out if we failed the conversion
        if (!success)
        {
            error = ilGetError();
            std::cout << "Image conversion failed - IL reports error: " << error << " - " << iluErrorString(error) << std::endl;
            exit(-1);
        }

        // Generate a new texture
        glGenTextures(1, &textureID);

        // Bind the texture to a name
        glBindTexture(GL_TEXTURE_2D, textureID);

        // Set texture clamping method
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);

        // Set texture interpolation method to use linear interpolation (no MIPMAPS)
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);

        // Specify the texture specification
        glTexImage2D(GL_TEXTURE_2D, 				// Type of texture
                0,				// Pyramid level (for mip-mapping) - 0 is the top level
                ilGetInteger(IL_IMAGE_FORMAT),	// Internal pixel format to use. Can be a generic type like GL_RGB or GL_RGBA, or a sized type
                ilGetInteger(IL_IMAGE_WIDTH),	// Image width
                ilGetInteger(IL_IMAGE_HEIGHT),	// Image height
                0,				// Border width in pixels (can either be 1 or 0)
                ilGetInteger(IL_IMAGE_FORMAT),	// Format of image pixel data
                GL_UNSIGNED_BYTE,		// Image data type
                ilGetData());			// The actual image data itself
    }
    else // If we failed to open the image file in the first place...
    {
        error = ilGetError();
        std::cout << "Image load failed - IL reports error: " << error << " - " << iluErrorString(error) << std::endl;
        exit(-1);
    }

    ilDeleteImages(1, &imageID); // Because we have already copied image data into texture data we can release memory used by image.

    std::cout << "Texture creation successful." << std::endl;

    tex1 =  textureID; // Return the GLuint to the texture so you can use it!
}

void Points::runKernelsPCISPH()
{
    hipMemset(dDensSum, 0, sizeof(float));
    hipMemset(dDensCount, 0, sizeof(int));

    if (numPoints == 0)
        return;
    //get pointers
    for (int i=0; i<BUFFER_COUNT; i++)
    {
        gpuErrchk(hipGraphicsMapResources(1, &(resources[i])));
        gpuErrchk(hipGraphicsResourceGetMappedPointer( &(dPointers[i]), &(numBytes[i]), resources[i]));
    }
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    if (!isPing)
    {
        //swap ping pong pointers
        Points::swapPtr(dPointers[BUFFER_STATES_PING], dPointers[BUFFER_STATES_PONG]);
        Points::swapPtr(dPointers[BUFFER_CELLID_PING], dPointers[BUFFER_CELLID_PONG]);
        Points::swapPtr(dPointers[BUFFER_POSITIONS_PING], dPointers[BUFFER_POSITIONS_PONG]);
        Points::swapPtr(dPointers[BUFFER_VELOCITIES_PING], dPointers[BUFFER_VELOCITIES_PONG]);
        Points::swapPtr(dPointers[BUFFER_TEMP_PING], dPointers[BUFFER_TEMP_PONG]);
        Points::swapPtr(dPointers[BUFFER_COLORGRAD_PING], dPointers[BUFFER_COLORGRAD_PONG]);
        Points::swapPtr(dPointers[BUFFER_DENSITIES_PING], dPointers[BUFFER_DENSITIES_PONG]);
    }
    resetGrid<<<16*16, 16>>>
        (grid.count,
         grid.gridRes);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk(hipMemcpy(dNumPoints, &numPoints, sizeof(uint), hipMemcpyHostToDevice));

    insertPars<<<max((numPoints-1)/1024+1, 1), 1024>>>
        ((float3*)dPointers[BUFFER_POSITIONS_PING],
         (uint*)dPointers[BUFFER_CELLID_PING],
         dNumPoints,
         (uint*)dPointers[BUFFER_OFFSET],
         grid.startPoint,
         grid.cellSize,
         grid.gridRes,
         grid.count,
         numPoints);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    prefixSumKernel<<<2,1024>>>(grid.count,grid.firstIdx,(uint*)dSumArray, grid.gridRes.x*grid.gridRes.y*grid.gridRes.z);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    prefexSumPostShort<<<2, 1024>>>(grid.firstIdx, (uint*)dSumArray, 2);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    countingSort<<<max((numPoints-1)/1024+1, 1), 1024>>>(
            grid.firstIdx,
            (uint*)dPointers[BUFFER_OFFSET],
            (uint*)dPointers[BUFFER_CELLID_PING],
            (uint*)dPointers[BUFFER_CELLID_PONG],
            grid.count,

            (float3*)dPointers[BUFFER_VELOCITIES_PING],
            (float3*)dPointers[BUFFER_VELOCITIES_PONG],
            (float3*)dPointers[BUFFER_POSITIONS_PING],
            (float3*)dPointers[BUFFER_POSITIONS_PONG],
            (States*)dPointers[BUFFER_STATES_PING],
            (States*)dPointers[BUFFER_STATES_PONG],
            (float*)dPointers[BUFFER_TEMP_PING],
            (float*)dPointers[BUFFER_TEMP_PONG],
            numPoints);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk(hipMemcpy(&numPoints, dNumPoints, sizeof(uint), hipMemcpyDeviceToHost));
    if (numPoints != 0)
    {
        calculateDensitiesGlobal<<<max((numPoints-1)/1024+1, 1), 1024>>>(
                (float3*)dPointers[BUFFER_POSITIONS_PONG], 
                (float*)dPointers[BUFFER_DENSITIES_PONG], 
                (uint*)dPointers[BUFFER_CELLID_PONG],
                grid.firstIdx,
                grid.count,
                numPoints);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        //compute external forces
    }
    for (int i=0; i<BUFFER_COUNT; i++)
        gpuErrchk(hipGraphicsUnmapResources(1, &(resources[i])));
}
void Points::runKernels()
{
    if (numPoints == 0)
        return;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    //get pointers
    for (int i=0; i<BUFFER_COUNT; i++)
    {
        gpuErrchk(hipGraphicsMapResources(1, &(resources[i])));
        gpuErrchk(hipGraphicsResourceGetMappedPointer( &(dPointers[i]), &(numBytes[i]), resources[i]));
    }
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    ImGui::Text("Num of Points\t %d", numPoints);
    ImGui::Text("Mapping\t %f",elapsedTime);

    /*ping:
      ping -sort-> pong -> render
      pong
      pong -sort-> ping -> render
      */

    if (!isPing)
    {
        //swap ping pong pointers
        Points::swapPtr(dPointers[BUFFER_STATES_PING], dPointers[BUFFER_STATES_PONG]);
        Points::swapPtr(dPointers[BUFFER_CELLID_PING], dPointers[BUFFER_CELLID_PONG]);
        Points::swapPtr(dPointers[BUFFER_POSITIONS_PING], dPointers[BUFFER_POSITIONS_PONG]);
        Points::swapPtr(dPointers[BUFFER_VELOCITIES_PING], dPointers[BUFFER_VELOCITIES_PONG]);
        Points::swapPtr(dPointers[BUFFER_TEMP_PING], dPointers[BUFFER_TEMP_PONG]);
        Points::swapPtr(dPointers[BUFFER_COLORGRAD_PING], dPointers[BUFFER_COLORGRAD_PONG]);
        Points::swapPtr(dPointers[BUFFER_DENSITIES_PING], dPointers[BUFFER_DENSITIES_PONG]);
    }
   
    hipEventRecord(start, 0);
    resetGrid<<<16*16, 16>>>
        (grid.count,
         grid.gridRes);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    ImGui::Text("Reset Grid\t %f",elapsedTime);

    //insert particles and count
    //std::cout<<"Inserting particles...\n";
    gpuErrchk(hipMemcpy(dNumPoints, &numPoints, sizeof(uint), hipMemcpyHostToDevice));


    hipEventRecord(start, 0);
    insertPars<<<max((numPoints-1)/1024+1, 1), 1024>>>
        ((float3*)dPointers[BUFFER_POSITIONS_PING],
         (uint*)dPointers[BUFFER_CELLID_PING],
         dNumPoints,
         (uint*)dPointers[BUFFER_OFFSET],
         grid.startPoint,
         grid.cellSize,
         grid.gridRes,
         grid.count,
         numPoints);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    ImGui::Text("Insert Pars\t %f",elapsedTime);

    //do prefix sum on particles
    hipEventRecord(start, 0);
    prefixSumKernel<<<2,1024>>>(grid.count,grid.firstIdx,(uint*)dSumArray, grid.gridRes.x*grid.gridRes.y*grid.gridRes.z);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    prefexSumPostShort<<<2, 1024>>>(grid.firstIdx, (uint*)dSumArray, 2);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );


    //std::cout<<"counting sort...\n";
    countingSort<<<max((numPoints-1)/1024+1, 1), 1024>>>(
            grid.firstIdx,
            (uint*)dPointers[BUFFER_OFFSET],
            (uint*)dPointers[BUFFER_CELLID_PING],
            (uint*)dPointers[BUFFER_CELLID_PONG],
            grid.count,

            (float3*)dPointers[BUFFER_VELOCITIES_PING],
            (float3*)dPointers[BUFFER_VELOCITIES_PONG],
            (float3*)dPointers[BUFFER_POSITIONS_PING],
            (float3*)dPointers[BUFFER_POSITIONS_PONG],
            (States*)dPointers[BUFFER_STATES_PING],
            (States*)dPointers[BUFFER_STATES_PONG],
            (float*)dPointers[BUFFER_TEMP_PING],
            (float*)dPointers[BUFFER_TEMP_PONG],
            numPoints);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    ImGui::Text("Counting sort\t %f",elapsedTime);

    //std::cout<<"Copying data back...\n";
    gpuErrchk(hipMemcpy(&numPoints, dNumPoints, sizeof(uint), hipMemcpyDeviceToHost));


    if (numPoints != 0)
    {
        //===============++Densities++=====================
        //std::cout<<"Calculating densities... numOfPar: "<<max((numPoints-1)/1024+1, 1)<<std::endl;;
        hipEventRecord(start, 0);
        calculateDensitiesGlobal<<<max((numPoints-1)/1024+1, 1), 1024>>>(
                (float3*)dPointers[BUFFER_POSITIONS_PONG], 
                (float*)dPointers[BUFFER_DENSITIES_PONG], 
                (uint*)dPointers[BUFFER_CELLID_PONG],
                grid.firstIdx,
                grid.count,
                numPoints);
        //calculateDensitiesShared<<<16*16*16, 100>>>(
        //        (float3*)dPointers[BUFFER_POSITIONS_PONG], 
        //        (float*)dPointers[BUFFER_DENSITIES], 
        //        (uint*)dPointers[BUFFER_CELLID_PONG],
        //        grid.firstIdx,
        //        grid.count,
        //        numPoints);

        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        ImGui::Text("Densities\t %f",elapsedTime);

        //Color grad
        hipEventRecord(start, 0);
            computeColorGradsGlobal<<<max((numPoints-1)/1024+1, 1), 1024>>>(
                    (float3*)dPointers[BUFFER_POSITIONS_PONG],
                    (float*)dPointers[BUFFER_DENSITIES_PONG],
                    (float3*)dPointers[BUFFER_COLORGRAD_PONG],
                    (float*)dPointers[BUFFER_TEMP_PONG],
                    (float*)dDensSum,
                    (uint*)dDensCount,
                    (uint*)dPointers[BUFFER_STATES_PONG],
                    (uint*)dPointers[BUFFER_CELLID_PONG],
                    grid.firstIdx,
                    grid.count,
                    numPoints
                    );
            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        ImGui::Text("Color Grad\t %f",elapsedTime);



        //=============++Forces++=======================
        //std::cout<<"Computing forces...\n";
#define FORCE_BLOCK_DIM 512
        hipEventRecord(start, 0);
        calculateForcesGlobal<<<max((numPoints-1)/FORCE_BLOCK_DIM+1,1), FORCE_BLOCK_DIM>>>(
                (float3*)dPointers[BUFFER_POSITIONS_PONG],
                (float*)dPointers[BUFFER_DENSITIES_PONG],
                (float3*)dPointers[BUFFER_VELOCITIES_PONG],
                (float3*)dPointers[BUFFER_PRESSURES],
                (float3*)dPointers[BUFFER_COLORGRAD_PONG],
                (uint*)dPointers[BUFFER_CELLID_PONG],
                (uint*)dPointers[BUFFER_STATES_PONG],
                (uint*)grid.firstIdx, 
                (uint*)grid.count,
                (float3*)dPointers[BUFFER_NORMALS],
                numPoints);

        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        ImGui::Text("Forces\t %f",elapsedTime);
    }
    //Unmap pointers;
    for (int i=0; i<BUFFER_COUNT; i++)
        gpuErrchk(hipGraphicsUnmapResources(1, &(resources[i])));
}
void Points::insertParsFromOBJ(std::string fileName, float temperature, GLuint state)
{
    std::cout<<"Inserting OBJ "<<fileName<<std::endl;
    std::vector<GLfloat> vertices;
    std::vector<GLfloat> vals;
    std::vector<GLuint> states;
    std::vector<GLfloat> velos;
    std::string str;
    std::ifstream inFile(fileName);

    while(std::getline(inFile, str))
    {
        splitstring split(str);
        vector<std::string> splited = split.split(' ');
        if (splited.size() > 0)
            if (splited[0] == "v")
                for (int i=1; i<splited.size(); i++)
                    vertices.push_back(std::stof(splited[i]));
    }
    for (auto i=0; i<vertices.size()/3; i++)
    {
        vals.push_back(temperature);
        states.push_back(state);
        velos.push_back(0.0);
        velos.push_back(0.0);
        velos.push_back(0.0);
    }
    std::cout<<states.size()<<std::endl;;
    std::cout<<"Num vertices: "<<vertices.size()<<std::endl;
    addPars(&vertices[0], &vals[0], &velos[0], states.size(), &states[0]);
}

void Points::insertBowl(const glm::vec3 &center, const glm::vec3 &dir)
{
    const float radius = 1.0;
    const uint X=16;
    const uint Y=16;
    const uint Z=16;
    const float step = 10.0/32.0;

    std::vector<GLfloat> pos;
    std::vector<GLfloat> vals;
    std::vector<GLuint> states;
    std::vector<GLfloat> velos;
    pos.reserve(X*Y*Z*3);
    vals.reserve(X*Y*Z);
    states.reserve(X*Y*Z);
    velos.reserve(X*Y*Z*3);

    GLuint numPar = 0;
    for (uint i=0; i<X; i++)
        for (uint j=0; j<Y; j++)
            for (uint k=0; k<Z; k++)
            {
                float r = (step*i-2.5)*(step*i-2.5)+
                    (step*j-2.5)*(step*j-2.5)+
                    (step*k-2.5)*(step*k-2.5);
                if ( 
                        r < 
                        radius * radius
                        && 
                        r > radius * radius -0.5 
                         &&
                         step*j-2.5 < 0.0
                   ){
                    pos.push_back(center.x-2.5 + step * i);
                    pos.push_back(center.y-2.5 + step * j);
                    pos.push_back(center.z-2.5 + step * k);
                    vals.push_back(80.0);
                    states.push_back(0);
                    velos.push_back(dir.x*200);
                    velos.push_back(dir.y*200);
                    velos.push_back(dir.z*200);
                    numPar++;
                }
            }
    addPars(&pos[0], &vals[0], &velos[0], numPar, &states[0]);
}

void Points::insertCUBE(const glm::vec3 &center, const glm::vec3 &dir)
{
    //init a grid or pars
    const uint X=20;
    const uint Y=20;
    const uint Z=20;
    float step = 10.0/16.0;
    GLfloat pos[X*Y*Z*3];
    GLfloat vals[X*Y*Z];
    GLuint states[X*Y*Z];
    GLfloat velos[X*Y*Z*3];
    uint parIdx = 0;
    for (uint i=0; i<X; i++)
        for (uint j=0; j<Y; j++)
            for (uint k=0; k<Z; k++)
            {
                pos[parIdx++] = center.x-7 + step * i;
                pos[parIdx++] = center.y-14 + step * j;
                pos[parIdx++] = center.z-14 + step * k;
            }
    for (uint i=0; i<X*Y*Z; i++)
    {
        vals[i] = 80.0;
        states[i] = 0;
        velos[3*i+0] = dir.x*100;
        velos[3*i+1] = dir.y*100;
        velos[3*i+2] = dir.z*100;
    }
    addPars(pos, vals,velos, X*Y*Z, states);
}

void Points::insertCUBEE()
{
    //init a grid or pars
    const uint X=20;
    const uint Y=4;
    const uint Z=20;
    float step = 10.0/26.0;
    GLfloat pos[X*Y*Z*3];
    GLfloat vals[X*Y*Z];
    GLuint states[X*Y*Z];
    GLfloat velos[X*Y*Z*3];
    uint parIdx = 0;
    for (uint i=0; i<X; i++)
        for (uint j=0; j<Y; j++)
            for (uint k=0; k<Z; k++)
            {
                pos[parIdx++] = -20 + step * i;
                pos[parIdx++] = -20 + step * k;
                pos[parIdx++] = -10 + step * j;
            }
    for (uint i=0; i<X*Y*Z; i++)
    {
        vals[i] = 10.0;
        states[i] = 0;
        velos[3*i+0] = 0.0;
        velos[3*i+1] = 0.0;
        velos[3*i+2] = 0.0;
    }
    addPars(pos, vals,velos, X*Y*Z, states);
}

void Points::insertOne(const glm::vec3& origin, const glm::vec3 &velocity, float temperature)
{
    addPars(&origin[0], &temperature, &velocity[0], 1, 0);
}

void Points::insertBar()
{
    //init a grid or pars
    const uint X=30;
    const uint Y=4;
    const uint Z=64;
    float step = 10.0/32.0;
    GLfloat pos[2*X*Y*Z*3];
    GLfloat vals[2*X*Y*Z];
    GLuint states[2*X*Y*Z];
    GLfloat velos[2*X*Y*Z*3];
    uint parIdx = 0;
    for (uint i=0; i<X; i++)
        for (uint j=0; j<Y; j++)
            for (uint k=0; k<Z; k++)
            {
                pos[parIdx++] = -5.2+ step * i;
                pos[parIdx++] = -10 + step * j;
                pos[parIdx++] = -15 + step * k;

                pos[parIdx++] = 5.2- step * i;
                pos[parIdx++] = -10 + step * j;
                pos[parIdx++] = -15 + step * k;
            }
    for (uint i=0; i<X*Y*Z*2; i++)
    {
        vals[i] = 30.0;
        states[i] = 1;
        velos[3*i+0] = 0.0;
        velos[3*i+1] = 0.0;
        velos[3*i+2] = 0.0;
    }
    addPars(pos, vals,velos, X*Y*Z, states);
}
